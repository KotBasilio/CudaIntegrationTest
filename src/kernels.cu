﻿#include <stdio.h>

// redundant -- to make intellisense work
#ifndef __CUDACC__
#define __CUDACC__
#endif

#ifdef __HIPCC__
#include "hip/hip_runtime.h"


__global__ void addKernel(int *c, const int *a, const int *b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

__host__ __device__ void SumParts(int* c, const int* a, const int* b)
{
   *c = *a + *b;
}

__global__ void addOtherKernel(int *c, const int *a, const int *b)
{
   int i = threadIdx.x;
   SumParts(c+i, a+i, b+i);
   c[i] += a[i] - 1;
}

__global__ void IncKernelFunc(int *c)
{
   int i = threadIdx.x;
   c[i]++;
}

// using CUDA to add vectors in parallel.
int addWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
   size_t freeMem, totalMem;
   int *dev_a = 0;
   int *dev_b = 0;
   int *dev_c = 0;
   auto alcSize = size * sizeof(int) * 1024 * 1024;
   int st;//cudaError_t st;

   // Choose which GPU to run on, change this on a multi-GPU system.
   st = hipSetDevice(0);
   if (st != hipSuccess) {
      fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
      goto Error;
   }

   // Allocate GPU buffers for three vectors (two input, one output)    .
   st = hipMalloc((void**)&dev_c, alcSize) 
      + hipMalloc((void**)&dev_a, alcSize) 
      + hipMalloc((void**)&dev_b, alcSize);
   if (st != hipSuccess) {
      fprintf(stderr, "hipMalloc failed!");
      goto Error;
   }

   st = hipMemGetInfo(&freeMem, &totalMem);
   if (st == hipSuccess) {
      printf("    after allocs : %.2f MB\n", freeMem / (1024.0 * 1024.0));
   }

   // Copy input vectors from host memory to GPU buffers.
   st = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
   if (st != hipSuccess) {
      fprintf(stderr, "hipMemcpy failed!");
      goto Error;
   }

   st = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
   if (st != hipSuccess) {
      fprintf(stderr, "hipMemcpy failed!");
      goto Error;
   }

   // Launch a kernel on the GPU with one thread for each element.
   //addKernel<<<1, size>>>(dev_c, dev_a, dev_b);
   addOtherKernel<<<1, size>>>(dev_c, dev_a, dev_b);
   //IncKernelFunc<<<1, size>>>(dev_c);
   //IncExtKernel<<<1, size>>>(dev_c);

   // Check for any errors launching the kernel
   st = hipGetLastError();
   if (st != hipSuccess) {
      fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString((hipError_t)st));
      goto Error;
   }

   st = hipMemGetInfo(&freeMem, &totalMem);
   if (st == hipSuccess) {
      printf("    after kernels: %.2f MB\n", freeMem / (1024.0 * 1024.0));
   }

   // cudaDeviceSynchronize waits for the kernel to finish, and returns
   // any errors encountered during the launch.
   st = hipDeviceSynchronize();
   if (st != hipSuccess) {
      fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", st);
      goto Error;
   }

   st = hipMemGetInfo(&freeMem, &totalMem);
   if (st == hipSuccess) {
      printf("    after sync   : %.2f MB\n", freeMem / (1024.0 * 1024.0));
   }

   // Copy output vector from GPU buffer to host memory.
   st = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
   if (st != hipSuccess) {
      fprintf(stderr, "hipMemcpy failed!");
      goto Error;
   }

Error:
   hipFree(dev_c);
   hipFree(dev_a);
   hipFree(dev_b);
    
   return st;
}

// Helper function for calculating core numbers based on version
int getCudaCoresPerSM(int major, int minor) {
   if (major == 8 && minor == 6) return 128; // Ampere (RTX 30xx)
   if (major == 8 && minor == 0) return 64;  // Ampere (A100)
   if (major == 7 && minor == 5) return 64;  // Turing (T4)
   if (major == 7 && minor == 0) return 64;  // Volta  (V100)
   if (major == 6 && minor == 1) return 128; // Pascal (P100)
   return -1; // unknonw
}

bool DetectCUDA()
{
   int deviceCount;
   hipGetDeviceCount(&deviceCount);
   printf("Available GPUs: %d\n", deviceCount);
   if (!deviceCount) {
      return false;
   }

   for (int i = 0; i < deviceCount; ++i) {
      hipDeviceProp_t deviceProp;
      hipGetDeviceProperties(&deviceProp, i);
      printf("Device %d: %s\n", i, deviceProp.name);
   }

   {
      int device;
      hipGetDevice(&device);

      hipDeviceProp_t prop;
      hipGetDeviceProperties(&prop, device);

      int coresPerSM = getCudaCoresPerSM(prop.major, prop.minor);
      int totalCores = prop.multiProcessorCount * coresPerSM;

      printf("GPU name: %s\n", prop.name);
      printf("Streaming Multiprocessors (SM): %d\n", prop.multiProcessorCount);
      if (totalCores > 0) {
         printf("CUDA Cores per SM: %d\n", coresPerSM);
         printf("Total CUDA Cores : %d\n", totalCores);
      } else {
         printf("CUDA Cores per SM is not recognized: version %d.%d\n", prop.major, prop.minor);
      }
   }

   size_t freeMem, totalMem;
   {
      hipError_t status = hipMemGetInfo(&freeMem, &totalMem);
      if (status != hipSuccess) {
         fprintf(stderr, "Error: %s\n", hipGetErrorString(status));
         return false;
      }

      printf("GPU memory total : %.2f MB\n", totalMem / (1024.0 * 1024.0));
      printf("       available : %.2f MB\n", freeMem / (1024.0 * 1024.0));
   }

   return true;
}

bool CudaWork(int* c, const int* a, const int* b, unsigned int size)
{
   DetectCUDA();

   // Add vectors in parallel.
   auto st = addWithCuda(c, a, b, size);
   if (st) {
      fprintf(stderr, "addWithCuda failed!");
      return false;
   }

   return true;
}

bool CudaClose()
{
   // cudaDeviceReset must be called before exiting in order for profiling and
   // tracing tools such as Nsight and Visual Profiler to show complete traces.
   auto st = hipDeviceReset();
   if (st != hipSuccess) {
      fprintf(stderr, "hipDeviceReset failed!");
      return false;
   }

   // final stat
   printf("CUDA reset the device; ");
   size_t freeMem, totalMem;
   st = hipMemGetInfo(&freeMem, &totalMem);
   if (st == hipSuccess) {
      printf("memory after reset : %.2f MB\n", freeMem / (1024.0 * 1024.0));
   }

   return true;
}

#else // Code stubs, compiled when CUDA is not available

bool DetectCUDA()
{
   fprintf(stderr, "CUDA didn't even compile. Surely it's unavailable on this platform.\nIt's very sad to work without CUDA.\n");
   return false;
}

bool CudaWork(int* c, const int* a, const int* b, unsigned int size)
{
   return false;
}

bool CudaClose()
{
   return true;
}

#endif

