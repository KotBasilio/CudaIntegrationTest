#include "hip/hip_runtime.h"
﻿#define  _CRT_SECURE_NO_WARNINGS

#include <stdio.h>
#include <malloc.h>
#include <string.h>

#include "TestSuite.h"

#include "hip/hip_runtime.h"
#include ""

void CTestSuite::CarpenterSolve()
{
   CarpTest();
}

#include "hip/hip_runtime.h"
#include ""

extern __global__ void kerCarpTest(void);

void CarpTest()
{
   unsigned int size = 5;
   kerCarpTest<<<1, size>>>();
}

__global__ void kerCarpTest(void)
{
   int i = threadIdx.x;
   i++;
}

