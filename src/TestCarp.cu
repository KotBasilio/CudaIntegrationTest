#include "hip/hip_runtime.h"
﻿#define  _CRT_SECURE_NO_WARNINGS

#include <stdio.h>
#include <malloc.h>
#include <string.h>

#include "TestSuite.h"

#include "hip/hip_runtime.h"
#include ""

extern __global__ void kerCarpTest(void);

void CTestSuite::CarpenterSolve()
{
   unsigned int size = 5;
   kerCarpTest<<<1, size>>>();
}

__global__ void kerCarpTest(void)
{
   int i = threadIdx.x;
   i++;
}
