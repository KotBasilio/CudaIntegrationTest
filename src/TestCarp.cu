#include "hip/hip_runtime.h"
﻿#define  _CRT_SECURE_NO_WARNINGS

#include <stdio.h>
#include <malloc.h>
#include <string.h>
#include <cassert>

#include "TestSuite.h"

#include "hip/hip_runtime.h"
#include ""

void CTestSuite::CarpenterSolve()
{
   printf("Testing Carpenter()");
   bool isAllright = true;

   // prepare all boards
   static boards _chunkBoards;
   int idxToadd = 0;
   int threadBegin = MAX_THREADS_IN_TEST - 1;
   for (int threadIndex = threadBegin; threadIndex >= 0; threadIndex--) {
      deal dl;
      int handno = 0;
      for (; handno < TEST_NUM_EXAMP_PKG; handno++) {
         FillDeal(dl, handno);
         _chunkBoards.deals[idxToadd] = dl;
         _chunkBoards.target[idxToadd] = -1;
         _chunkBoards.solutions[idxToadd] = 3;
         _chunkBoards.mode[idxToadd] = 0;
         idxToadd++;

         _chunkBoards.deals[idxToadd] = dl;
         _chunkBoards.target[idxToadd] = -1;
         _chunkBoards.solutions[idxToadd] = 2;
         _chunkBoards.mode[idxToadd] = 0;
         idxToadd++;
      }

      for (; handno < TEST_HOLDINGS_COUNT; handno++) {
         FillDeal(dl, handno);
         _chunkBoards.deals[idxToadd] = dl;
         _chunkBoards.target[idxToadd] = -1;
         _chunkBoards.solutions[idxToadd] = 1;
         _chunkBoards.mode[idxToadd] = 0;
         idxToadd++;

         dl.trump = 0;
         dl.first = 0;
         _chunkBoards.deals[idxToadd] = dl;
         _chunkBoards.target[idxToadd] = -1;
         _chunkBoards.solutions[idxToadd] = 1;
         _chunkBoards.mode[idxToadd] = 0;
         idxToadd++;
      }
   }
   assert(idxToadd == TOTAL_FUTURES_IN_TEST);
   _chunkBoards.noOfBoards = idxToadd;

   // run with CUDA
   Carpenter carp;
   carp.SmallTest(_chunkBoards);

   // compare
   ControlSolvedBoards(isAllright);
}

extern __global__ void kerCarpTest(void);

__global__ void kerCarpTest(void)
{
   int i = threadIdx.x;
   i++;
}

Carpenter::Carpenter()
{
}

Carpenter::~Carpenter()
{
   printf("~");
}

void Carpenter::SmallTest(boards &chunk)
{
   printf("...");

   unsigned int size = MAX_THREADS_IN_TEST;
   kerCarpTest << <1, size >> > ();

}

