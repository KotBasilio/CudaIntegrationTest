#include "hip/hip_runtime.h"
﻿#define  _CRT_SECURE_NO_WARNINGS

#include <stdio.h>
#include <malloc.h>
#include <string.h>
#include <cassert>

#include "TestSuite.h"

void CTestSuite::PrepareChunk(boards& _chunkBoards)
{
   int idxToadd = 0;
   int threadBegin = MAX_THREADS_IN_TEST - 1;
   for (int threadIndex = threadBegin; threadIndex >= 0; threadIndex--) {
      deal dl;
      int handno = 0;
      for (; handno < TEST_NUM_EXAMP_PKG; handno++) {
         FillDeal(dl, handno);
         _chunkBoards.deals[idxToadd] = dl;
         _chunkBoards.target[idxToadd] = -1;
         _chunkBoards.solutions[idxToadd] = 3;
         _chunkBoards.mode[idxToadd] = 0;
         idxToadd++;

         _chunkBoards.deals[idxToadd] = dl;
         _chunkBoards.target[idxToadd] = -1;
         _chunkBoards.solutions[idxToadd] = 2;
         _chunkBoards.mode[idxToadd] = 0;
         idxToadd++;
      }

      for (; handno < TEST_HOLDINGS_COUNT; handno++) {
         FillDeal(dl, handno);
         _chunkBoards.deals[idxToadd] = dl;
         _chunkBoards.target[idxToadd] = -1;
         _chunkBoards.solutions[idxToadd] = 1;
         _chunkBoards.mode[idxToadd] = 0;
         idxToadd++;

         dl.trump = 0;
         dl.first = 0;
         _chunkBoards.deals[idxToadd] = dl;
         _chunkBoards.target[idxToadd] = -1;
         _chunkBoards.solutions[idxToadd] = 1;
         _chunkBoards.mode[idxToadd] = 0;
         idxToadd++;
      }
   }
   assert(idxToadd == TOTAL_FUTURES_IN_TEST);
   _chunkBoards.noOfBoards = idxToadd;
}

void CTestSuite::CarpenterSolve()
{
   printf("Testing Carpenter()");
   bool isAllright = true;

   // prepare all boards
   static boards _chunkBoards;
   PrepareChunk(_chunkBoards);

   // run with CUDA
   Carpenter carp;
   carp.SmallTest();
   carp.SolveChunk(_chunkBoards);

   // compare
   ControlSolvedBoards(isAllright);
}

__global__ void kerCarpTest(void)
{
   int i = threadIdx.x;
   i++;
}

void Carpenter::SmallTest()
{
   printf("...");
   unsigned int size = MAX_THREADS_IN_TEST;
   kerCarpTest << <1, size >> > ();
}

