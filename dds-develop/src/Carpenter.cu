#include "hip/hip_runtime.h"
/*
   2024 GPU acceleration by Serge Mironov
   requires CUDA technology
   its toolkit is available here:
   https://developer.nvidia.com/cuda-toolkit
*/

#include <iostream>
#include <iomanip>
#include <sstream>

#include "../include/dll.h"
#include "SolverIF.h"
#include "Init.h"
#include "ABsearch.h"
#include "System.h"
#include "Scheduler.h"

#include "LogSubsys.cu"

void CopyToDeviceConstants()
{
   //hipMemcpyToSymbol(HIP_SYMBOL(d_highestRank), highestRank, sizeof(highestRank));
   //hipMemcpyToSymbol(HIP_SYMBOL(d_lowestRank), lowestRank, sizeof(lowestRank));
   //hipMemcpyToSymbol(HIP_SYMBOL(d_counttable), counttable, sizeof(counttable));
   //hipMemcpyToSymbol(HIP_SYMBOL(d_relRank), relRank, sizeof(relRank));
   //hipMemcpyToSymbol(HIP_SYMBOL(d_winRanks), winRanks, sizeof(winRanks));
   //hipMemcpyToSymbol(HIP_SYMBOL(d_groupData), groupData, sizeof(groupData));
   //hipMemcpyToSymbol(HIP_SYMBOL(d_bitMapRank), bitMapRank, sizeof(bitMapRank));
}

class CarpImpl {
   System sysdep;
   //Memory memory;
   //Scheduler scheduler;

public:
   CarpImpl() {
      // LogSubsystem is __managed__, so we initialize it explicitly
      //hipDeviceSynchronize();
      myLog.Initialize();
   }

   ~CarpImpl() {
      // Print the log on destruction
      hipDeviceSynchronize();
      myLog.PrintLog();
   }
};

Carpenter::Carpenter()
{
   Instance = new CarpImpl();
}

Carpenter::~Carpenter()
{
   if (Instance) {
      delete Instance;
      Instance = nullptr;
   }
}

__device__ int Carpenter::SolveBoard(const deal& dl, const int target, const int solutions, const int mode, futureTricks* futp, ThreadData* thrp)
{
   return 1;
}

__global__ void CarpFanOut(Carpenter * carp, boards & chunk)
{
   int i = threadIdx.x;
   if (i == 163) {
      LOG(SUCCESS);
   }
   //deal* myDeal = chunk.deals + i;
   //carp->SolveBoard(
   //   chunk.deals[i],
   //   chunk.target[i],
   //   chunk.solutions[i],
   //   chunk.mode[i],
   //   nullptr,
   //   nullptr
   //);
}

void Carpenter::SolveChunk(boards& chunk)
{
   printf("...");
   CarpFanOut << <1, chunk.noOfBoards >> > (this, chunk);
}

